#include "hip/hip_runtime.h"
#include <assert.h>

#include "cuZFP.h"

#include "encode1.cuh"
#include "encode2.cuh"
#include "encode3.cuh"

#include "decode1.cuh"
#include "decode2.cuh"
#include "decode3.cuh"

#include "ErrorCheck.h"

#include "constant_setup.cuh"
#include "pointers.cuh"
#include "type_info.cuh"
#include <iostream>
#include <assert.h>

// we need to know about bitstream, but we don't 
// want duplicate symbols.
#ifndef inline_
  #define inline_ inline
#endif

#include "../inline/bitstream.c"
namespace internal 
{ 
  
bool is_contigous3d(const uint dims[3], const int3 &stride, long long int &offset)
{
  typedef long long int int64;
  int64 idims[3];
  idims[0] = dims[0];
  idims[1] = dims[1];
  idims[2] = dims[2];

  int64 imin = std::min(stride.x,0) * (idims[0] - 1) + 
               std::min(stride.y,0) * (idims[1] - 1) + 
               std::min(stride.z,0) * (idims[2] - 1);

  int64 imax = std::max(stride.x,0) * (idims[0] - 1) + 
               std::max(stride.y,0) * (idims[1] - 1) + 
               std::max(stride.z,0) * (idims[2] - 1);
  offset = imin;
  int64 ns = idims[0] * idims[1] * idims[2];

  return (imax - imin + 1 == ns);
}

bool is_contigous2d(const uint dims[3], const int3 &stride, long long int &offset)
{
  typedef long long int int64;
  int64 idims[2];
  idims[0] = dims[0];
  idims[1] = dims[1];

  int64 imin = std::min(stride.x,0) * (idims[0] - 1) + 
               std::min(stride.y,0) * (idims[1] - 1);

  int64  imax = std::max(stride.x,0) * (idims[0] - 1) + 
                std::max(stride.y,0) * (idims[1] - 1); 

  offset = imin;
  return (imax - imin + 1) == (idims[0] * idims[1]);
}

bool is_contigous1d(uint dim, const int &stride, long long int &offset)
{
  offset = 0;
  if(stride < 0) offset = stride * (int(dim) - 1);
  return std::abs(stride) == 1;
}

bool is_contigous(const uint dims[3], const int3 &stride, long long int &offset)
{
  int d = 0;
  
  if(dims[0] != 0) d++;
  if(dims[1] != 0) d++;
  if(dims[2] != 0) d++;

  if(d == 3)
  {
    return is_contigous3d(dims, stride, offset);
  }
  else if(d == 2)
  {
   return is_contigous2d(dims, stride, offset);
  }
  else
  {
    return is_contigous1d(dims[0], stride.x, offset);
  } 

}
//
// encode expects device pointers
//
template<typename T>
size_t encode(uint dims[3], int3 stride, int bits_per_block, T *d_data, Word *d_stream)
{
  int d = 0;
  size_t len = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      d++;
      len *= dims[i];
    }
  }

  ErrorCheck errors;
  size_t stream_size = 0;
  if(d == 1)
  {
    int dim = dims[0];
    int sx = stride.x;
    gettimeofday(&cuda_start20S, NULL);
    cuZFP::ConstantSetup::setup_1d();
    gettimeofday(&cuda_start20E, NULL);
    cuda20 = ((cuda_start20E.tv_sec*1000000+cuda_start20E.tv_usec)-(cuda_start20S.tv_sec*1000000+cuda_start20S.tv_usec))/1000000.0;

    gettimeofday(&cuda_start21S, NULL);
    stream_size = cuZFP::encode1<T>(dim, sx, d_data, d_stream, bits_per_block); // kernel run time 
    gettimeofday(&cuda_start21E, NULL);
    cuda21 = ((cuda_start21E.tv_sec*1000000+cuda_start21E.tv_usec)-(cuda_start21S.tv_sec*1000000+cuda_start21S.tv_usec))/1000000.0;
  }
  else if(d == 2)
  {
    uint2 ndims = make_uint2(dims[0], dims[1]);
    int2 s;
    s.x = stride.x; 
    s.y = stride.y; 
    cuZFP::ConstantSetup::setup_2d();
    stream_size = cuZFP::encode2<T>(ndims, s, d_data, d_stream, bits_per_block); 
  }
  else if(d == 3)
  {
    int3 s;
    s.x = stride.x; 
    s.y = stride.y; 
    s.z = stride.z; 
    uint3 ndims = make_uint3(dims[0], dims[1], dims[2]);
    cuZFP::ConstantSetup::setup_3d();
    stream_size = cuZFP::encode<T>(ndims, s, d_data, d_stream, bits_per_block); 
  }

  errors.chk("Encode");
  
  return stream_size; 
}

template<typename T>
size_t decode(uint ndims[3], int3 stride, int bits_per_block, Word *stream, T *out)
{

  int d = 0;
  size_t out_size = 1;
  size_t stream_bytes = 0;
  for(int i = 0; i < 3; ++i)
  {
    if(ndims[i] != 0)
    {
      d++;
      out_size *= ndims[i];
    }
  }

  if(d == 3)
  {
    uint3 dims = make_uint3(ndims[0], ndims[1], ndims[2]);

    int3 s;
    s.x = stride.x; 
    s.y = stride.y; 
    s.z = stride.z; 

    cuZFP::ConstantSetup::setup_3d();
    stream_bytes = cuZFP::decode3<T>(dims, s, stream, out, bits_per_block); 
  }
  else if(d == 1)
  {
    uint dim = ndims[0];
    int sx = stride.x;

    cuZFP::ConstantSetup::setup_1d();
    stream_bytes = cuZFP::decode1<T>(dim, sx, stream, out, bits_per_block); 

  }
  else if(d == 2)
  {
    uint2 dims;
    dims.x = ndims[0];
    dims.y = ndims[1];

    int2 s;
    s.x = stride.x; 
    s.y = stride.y; 

    cuZFP::ConstantSetup::setup_2d();
    stream_bytes = cuZFP::decode2<T>(dims, s, stream, out, bits_per_block); 
  }
  else std::cerr<<" d ==  "<<d<<" not implemented\n";
 
  return stream_bytes;
}

Word *setup_device_stream(zfp_stream *stream,const zfp_field *field)
{
  gettimeofday(&cuda_start110S, NULL);
  bool stream_device = cuZFP::is_gpu_ptr(stream->stream->begin);
  gettimeofday(&cuda_start110E, NULL);
  cuda110 = ((cuda_start110E.tv_sec*1000000+cuda_start110E.tv_usec)-(cuda_start110S.tv_sec*1000000+cuda_start110S.tv_usec))/1000000.0;

  assert(sizeof(word) == sizeof(Word)); // "CUDA version currently only supports 64bit words");

  if(stream_device)
  {
    return (Word*) stream->stream->begin;
  } 

  Word *d_stream = NULL;
  // TODO: we we have a real stream we can just ask it how big it is
  size_t max_size = zfp_stream_maximum_size(stream, field);

  printf("max_size=%u\n", max_size);
  gettimeofday(&cuda_start111S, NULL);
  hipMalloc(&d_stream, max_size); // zfp buffer size. 
  hipMemcpy(d_stream, stream->stream->begin, max_size, hipMemcpyHostToDevice);
  gettimeofday(&cuda_start111E, NULL);
  cuda111 = ((cuda_start111E.tv_sec*1000000+cuda_start111E.tv_usec)-(cuda_start111S.tv_sec*1000000+cuda_start111S.tv_usec))/1000000.0; 
  return d_stream;
}

void * offset_void(zfp_type type, void *ptr, long long int offset)
{
  void * offset_ptr = NULL;
  if(type == zfp_type_float)
  {
    float* data = (float*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  else if(type == zfp_type_double)
  {
    double* data = (double*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  else if(type == zfp_type_int32)
  {
    int * data = (int*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  else if(type == zfp_type_int64)
  {
    long long int * data = (long long int*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  return offset_ptr;
}

void *setup_device_field(const zfp_field *field, const int3 &stride, long long int &offset)
{

  gettimeofday(&cuda_start100S, NULL);
  bool field_device = cuZFP::is_gpu_ptr(field->data);
  gettimeofday(&cuda_start100E, NULL);
  cuda100 = ((cuda_start100E.tv_sec*1000000+cuda_start100E.tv_usec)-(cuda_start100S.tv_sec*1000000+cuda_start100S.tv_usec))/1000000.0;

  if(field_device)
  {
    offset = 0;
    return field->data; // return the pointer to the array data.
  }
  uint dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;

  size_t type_size = zfp_type_size(field->type);

  size_t field_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      field_size *= dims[i];
    }
  }

  bool contig = internal::is_contigous(dims, stride, offset);
  
  void * host_ptr = offset_void(field->type, field->data, offset);;

  void *d_data = NULL;
  if(contig)
  {
    size_t field_bytes = type_size * field_size;
    printf("field_bytes=%u\n",  field_bytes);
    gettimeofday(&cuda_start101S, NULL);
    hipMalloc(&d_data, field_bytes); // field_bytes=
    hipMemcpy(d_data, host_ptr, field_bytes, hipMemcpyHostToDevice);
    gettimeofday(&cuda_start101E, NULL);
    cuda101 = ((cuda_start101E.tv_sec*1000000+cuda_start101E.tv_usec)-(cuda_start101S.tv_sec*1000000+cuda_start101S.tv_usec))/1000000.0;

  }

  return offset_void(field->type, d_data, -offset);
}

void cleanup_device_ptr(void *orig_ptr, void *d_ptr, size_t bytes, long long int offset, zfp_type type)
{
  gettimeofday(&cuda_start30S, NULL);
  bool device = cuZFP::is_gpu_ptr(orig_ptr);
  gettimeofday(&cuda_start30E, NULL);
  cuda30 = ((cuda_start30E.tv_sec*1000000+cuda_start30E.tv_usec)-(cuda_start30S.tv_sec*1000000+cuda_start30S.tv_usec))/1000000.0;


  if(device)
  {
    return;
  }
  // from whence it came
  void *d_offset_ptr = offset_void(type, d_ptr, offset);
  void *h_offset_ptr = offset_void(type, orig_ptr, offset);

  if(bytes > 0)
  {
    gettimeofday(&cuda_start31S, NULL);
    hipMemcpy(h_offset_ptr, d_offset_ptr, bytes, hipMemcpyDeviceToHost);
    gettimeofday(&cuda_start31E, NULL);
    cuda31 = ((cuda_start31E.tv_sec*1000000+cuda_start31E.tv_usec)-(cuda_start31S.tv_sec*1000000+cuda_start31S.tv_usec))/1000000.0;
  }

  gettimeofday(&cuda_start32S, NULL);
  hipFree(d_offset_ptr);
  gettimeofday(&cuda_start32E, NULL);
  cuda32 = ((cuda_start32E.tv_sec*1000000+cuda_start32E.tv_usec)-(cuda_start32S.tv_sec*1000000+cuda_start32S.tv_usec))/1000000.0;
}

} // namespace internal

size_t
cuda_compress(zfp_stream *stream, const zfp_field *field)
{
  //jwang
  
  gettimeofday(&cuda_start1S, NULL); //cuda1
  uint dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;
  
  int3 stride;  
  stride.x = field->sx ? field->sx : 1;
  stride.y = field->sy ? field->sy : field->nx;
  stride.z = field->sz ? field->sz : field->nx * field->ny;
  
  size_t stream_bytes = 0;
  long long int offset = 0; 

  gettimeofday(&cuda_start10S, NULL); // cuda10
  void *d_data = internal::setup_device_field(field, stride, offset);
  gettimeofday(&cuda_start10E, NULL); // cuda10
  cuda10 = ((cuda_start10E.tv_sec*1000000+cuda_start10E.tv_usec)-(cuda_start10S.tv_sec*1000000+cuda_start10S.tv_usec))/1000000.0;

  if(d_data == NULL)
  {
    // null means the array is non-contiguous host mem which is not supported
    return 0;
  }
  gettimeofday(&cuda_start11S, NULL); // cuda11
  Word *d_stream = internal::setup_device_stream(stream, field);
  gettimeofday(&cuda_start11E, NULL); // cuda11
  cuda11 = ((cuda_start11E.tv_sec*1000000+cuda_start11E.tv_usec)-(cuda_start11S.tv_sec*1000000+cuda_start11S.tv_usec))/1000000.0;

  gettimeofday(&cuda_start1E, NULL); //cuda1
  gettimeofday(&cuda_start2S, NULL);
  
  if(field->type == zfp_type_float)
  {
    float* data = (float*) d_data;
    stream_bytes = internal::encode<float>(dims, stride, (int)stream->maxbits, data, d_stream); //
  }
  else if(field->type == zfp_type_double)
  {
    double* data = (double*) d_data;
    stream_bytes = internal::encode<double>(dims, stride, (int)stream->maxbits, data, d_stream); //
  }
  else if(field->type == zfp_type_int32)
  {
    int * data = (int*) d_data;
    stream_bytes = internal::encode<int>(dims, stride, (int)stream->maxbits, data, d_stream);
  }
  else if(field->type == zfp_type_int64)
  {
    long long int * data = (long long int*) d_data;
    stream_bytes = internal::encode<long long int>(dims, stride, (int)stream->maxbits, data, d_stream);
  }

  gettimeofday(&cuda_start2E, NULL); //cuda2
  gettimeofday(&cuda_start3S, NULL); //cuda3

  internal::cleanup_device_ptr(stream->stream->begin, d_stream, stream_bytes, 0, field->type);
  internal::cleanup_device_ptr(field->data, d_data, 0, offset, field->type);

  // zfp wants to flush the stream.
  // set bits to wsize because we already did that.
  size_t compressed_size = stream_bytes / sizeof(Word);
  stream->stream->bits = wsize;
  // set stream pointer to end of stream
  stream->stream->ptr = stream->stream->begin + compressed_size;
  gettimeofday(&cuda_start3E, NULL); //cuda3
  
  cuda1 = ((cuda_start1E.tv_sec*1000000+cuda_start1E.tv_usec)-(cuda_start1S.tv_sec*1000000+cuda_start1S.tv_usec))/1000000.0;
  cuda2 = ((cuda_start2E.tv_sec*1000000+cuda_start2E.tv_usec)-(cuda_start2S.tv_sec*1000000+cuda_start2S.tv_usec))/1000000.0;
  cuda3 = ((cuda_start3E.tv_sec*1000000+cuda_start3E.tv_usec)-(cuda_start3S.tv_sec*1000000+cuda_start3S.tv_usec))/1000000.0;

  return stream_bytes;
}


  
void 
cuda_decompress(zfp_stream *stream, zfp_field *field)
{
  uint dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;
   
  int3 stride;  
  stride.x = field->sx ? field->sx : 1;
  stride.y = field->sy ? field->sy : field->nx;
  stride.z = field->sz ? field->sz : field->nx * field->ny;

  size_t decoded_bytes = 0;
  long long int offset = 0;
  void *d_data = internal::setup_device_field(field, stride, offset);
  
  if(d_data == NULL)
  {
    // null means the array is non-contiguous host mem which is not supported
    return;
  }

  Word *d_stream = internal::setup_device_stream(stream, field);

  if(field->type == zfp_type_float)
  {
    float *data = (float*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_double)
  {
    double *data = (double*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_int32)
  {
    int *data = (int*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_int64)
  {
    long long int *data = (long long int*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else
  {
    std::cerr<<"Cannot decompress: type unknown\n";
  }

   
  size_t type_size = zfp_type_size(field->type);

  size_t field_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      field_size *= dims[i];
    }
  }
  
  size_t bytes = type_size * field_size;
  internal::cleanup_device_ptr(stream->stream->begin, d_stream, 0, 0, field->type);
  internal::cleanup_device_ptr(field->data, d_data, bytes, offset, field->type);
  
  // this is how zfp determins if this was a success
  size_t words_read = decoded_bytes / sizeof(Word);
  stream->stream->bits = wsize;
  // set stream pointer to end of stream
  stream->stream->ptr = stream->stream->begin + words_read;
}
